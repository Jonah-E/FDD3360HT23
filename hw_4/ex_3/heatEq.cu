#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <math.h>
#include <stdlib.h>
#include <sys/time.h>
#include <thrust/device_ptr.h>
#include <thrust/sequence.h>

#define gpuCheck(stmt)                                                         \
  do {                                                                         \
    hipError_t err = stmt;                                                    \
    if (err != hipSuccess) {                                                  \
      printf("ERROR. Failed to run stmt %s\n", #stmt);                         \
      break;                                                                   \
    }                                                                          \
  } while (0)

// Macro to check the cuBLAS status
#define cublasCheck(stmt)                                                      \
  do {                                                                         \
    hipblasStatus_t err = stmt;                                                 \
    if (err != HIPBLAS_STATUS_SUCCESS) {                                        \
      printf("ERROR. Failed to run cuBLAS stmt %s\n", #stmt);                  \
      break;                                                                   \
    }                                                                          \
  } while (0)

// Macro to check the cuSPARSE status
#define cusparseCheck(stmt)                                                    \
  do {                                                                         \
    hipsparseStatus_t err = stmt;                                               \
    if (err != HIPSPARSE_STATUS_SUCCESS) {                                      \
      printf("ERROR. Failed to run cuSPARSE stmt %s\n", #stmt);                \
      break;                                                                   \
    }                                                                          \
  } while (0)

struct timeval t_start, t_end;
void cputimer_start() { gettimeofday(&t_start, 0); }
void cputimer_stop(const char* info)
{
  gettimeofday(&t_end, 0);
  double time = (1000000.0 * (t_end.tv_sec - t_start.tv_sec) + t_end.tv_usec -
                 t_start.tv_usec);
  printf("Timing - %s. \t\tElasped %.0f microseconds \n", info, time);
}

// Initialize the sparse matrix needed for the heat time step
void matrixInit(double* A, int* ArowPtr, int* AcolIndx, int dimX, double alpha)
{
  // Stencil from the finete difference discretization of the equation
  double stencil[] = {1, -2, 1};
  // Variable holding the position to insert a new element
  size_t ptr = 0;
  // Insert a row of zeros at the beginning of the matrix
  ArowPtr[1] = ptr;
  // Fill the non zero entries of the matrix
  for (int i = 1; i < (dimX - 1); ++i) {
    // Insert the elements: A[i][i-1], A[i][i], A[i][i+1]
    for (int k = 0; k < 3; ++k) {
      // Set the value for A[i][i+k-1]
      A[ptr] = stencil[k];
      // Set the column index for A[i][i+k-1]
      AcolIndx[ptr++] = i + k - 1;
    }
    // Set the number of newly added elements
    ArowPtr[i + 1] = ptr;
  }
  // Insert a row of zeros at the end of the matrix
  ArowPtr[dimX] = ptr;
}

int main(int argc, char** argv)
{
  int device = 0;              // Device to be used
  int dimX;                    // Dimension of the metal rod
  int nsteps;                  // Number of time steps to perform
  double alpha = 0.4;          // Diffusion coefficient
  double* temp;                // Array to store the final time step
  double* A;                   // Sparse matrix A values in the CSR format
  int* ARowPtr;                // Sparse matrix A row pointers in the CSR format
  int* AColIndx;               // Sparse matrix A col values in the CSR format
  int nzv;                     // Number of non zero values in the sparse matrix
  double* tmp;                 // Temporal array of dimX for computations
  size_t bufferSize = 0;       // Buffer size needed by some routines
  void* buffer = nullptr;      // Buffer used by some routines in the libraries
  int concurrentAccessQ;       // Check if concurrent access flag is set
  double zero = 0;             // Zero constant
  double one = 1;              // One constant
  double norm;                 // Variable for norm values
  double error;                // Variable for storing the relative error
  double tempLeft = 200.;      // Left heat source applied to the rod
  double tempRight = 300.;     // Right heat source applied to the rod
  hipblasHandle_t cublasHandle; // cuBLAS handle
  hipsparseHandle_t cusparseHandle;  // cuSPARSE handle
  hipsparseSpMatDescr_t Adescriptor; // Mat descriptor needed by cuSPARSE
  hipsparseDnVecDescr_t tempDesc;
  hipsparseDnVecDescr_t tmpDesc;

  // Read the arguments from the command line
  dimX = atoi(argv[1]);
  nsteps = atoi(argv[2]);

  // Print input arguments
  printf("The X dimension of the grid is %d \n", dimX);
  printf("The number of time steps to perform is %d \n", nsteps);

  // Get if the hipDeviceAttributeConcurrentManagedAccess flag is set
  gpuCheck(hipDeviceGetAttribute(&concurrentAccessQ,
                                  hipDeviceAttributeConcurrentManagedAccess, device));

  // Calculate the number of non zero values in the sparse matrix. This number
  // is known from the structure of the sparse matrix
  nzv = 3 * dimX - 6;

  cputimer_start();
  gpuCheck(hipMallocManaged(&temp, sizeof(double) * dimX));
  gpuCheck(hipMallocManaged(&A, sizeof(double) * nzv));
  gpuCheck(hipMallocManaged(&ARowPtr, sizeof(double) * (dimX + 1)));
  gpuCheck(hipMallocManaged(&AColIndx, sizeof(double) * nzv));
  gpuCheck(hipMallocManaged(&tmp, sizeof(double) * dimX));
  cputimer_stop("Allocating device memory");

  // Check if concurrentAccessQ is non zero in order to prefetch memory
  if (concurrentAccessQ) {
    cputimer_start();
    hipGetDevice(&device);
    hipMemPrefetchAsync(temp, sizeof(double) * dimX, hipCpuDeviceId, NULL);
    hipMemPrefetchAsync(A, sizeof(double) * nzv, hipCpuDeviceId, NULL);
    hipMemPrefetchAsync(ARowPtr, sizeof(double) * (dimX + 1), hipCpuDeviceId,
                         NULL);
    hipMemPrefetchAsync(AColIndx, sizeof(double) * nzv, hipCpuDeviceId, NULL);
    hipMemPrefetchAsync(tmp, sizeof(double) * dimX, hipCpuDeviceId, NULL);
    hipDeviceSynchronize();
    cputimer_stop("Prefetching GPU memory to the host");
  }

  // Initialize the sparse matrix
  cputimer_start();
  matrixInit(A, ARowPtr, AColIndx, dimX, alpha);
  cputimer_stop("Initializing the sparse matrix on the host");

  // Initiliaze the boundary conditions for the heat equation
  cputimer_start();
  memset(temp, 0, sizeof(double) * dimX);
  temp[0] = tempLeft;
  temp[dimX - 1] = tempRight;
  cputimer_stop("Initializing memory on the host");

  if (concurrentAccessQ) {
    cputimer_start();
    hipGetDevice(&device);
    hipMemPrefetchAsync(temp, sizeof(double) * dimX, device, NULL);
    hipMemPrefetchAsync(A, sizeof(double) * nzv, device, NULL);
    hipMemPrefetchAsync(ARowPtr, sizeof(double) * (dimX + 1), device, NULL);
    hipMemPrefetchAsync(AColIndx, sizeof(double) * nzv, device, NULL);
    hipMemPrefetchAsync(tmp, sizeof(double) * dimX, device, NULL);
    hipDeviceSynchronize();
    cputimer_stop("Prefetching GPU memory to the device");
  }

  cublasCheck(hipblasCreate(&cublasHandle));

  cusparseCheck(hipsparseCreate(&cusparseHandle));

  cublasCheck(hipblasSetPointerMode(cublasHandle, HIPBLAS_POINTER_MODE_HOST));
  cusparseCheck(
      hipsparseSetPointerMode(cusparseHandle, HIPSPARSE_POINTER_MODE_HOST));

  cusparseCheck(hipsparseCreateCsr(&Adescriptor, dimX, dimX, nzv,
                                  (void*) ARowPtr, (void*) AColIndx, (void*) A,
                                  HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                  HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F));
  cusparseCheck(hipsparseCreateDnVec(&tempDesc, dimX, (void*) temp, HIP_R_64F));
  cusparseCheck(hipsparseCreateDnVec(&tmpDesc, dimX, (void*) tmp, HIP_R_64F));

  cusparseCheck(hipsparseSpMV_bufferSize(
      cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &one, Adescriptor,
      tempDesc, &zero, tmpDesc, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT,
      &bufferSize));
  gpuCheck(hipMalloc(&buffer, bufferSize));

  // Perform the time step iterations
  for (int it = 0; it < nsteps; ++it) {
    cusparseCheck(hipsparseSpMV(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                               &one, Adescriptor, tempDesc, &zero, tmpDesc,
                               HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, buffer));
    hipDeviceSynchronize();

    cublasCheck(hipblasDaxpy(cublasHandle, dimX, &alpha, tmp, 1, temp, 1));
    hipDeviceSynchronize();

    cublasCheck(hipblasDnrm2(cublasHandle, dimX, tmp, 1, &norm));
    hipDeviceSynchronize();

    // If the norm of A*temp is smaller than 10^-4 exit the loop
    if (norm < 1e-4)
      break;
  }

  // Calculate the exact solution using thrust
  thrust::device_ptr<double> thrustPtr(tmp);
  thrust::sequence(thrustPtr, thrustPtr + dimX, tempLeft,
                   (tempRight - tempLeft) / (dimX - 1));

  // Calculate the relative approximation error:
  cublasCheck(hipblasDnrm2(cublasHandle, dimX, tmp, 1, &norm));
  hipDeviceSynchronize();

  one = -1;
  cublasCheck(hipblasDaxpy(cublasHandle, dimX, &one, temp, 1, tmp, 1));
  hipDeviceSynchronize();

  cublasCheck(hipblasDnrm2(cublasHandle, dimX, tmp, 1, &norm));
  hipDeviceSynchronize();
  error = norm;

  cublasCheck(hipblasDnrm2(cublasHandle, dimX, temp, 1, &norm));
  hipDeviceSynchronize();

  // Calculate the relative error
  error = error / norm;
  printf("The relative error of the approximation is %f\n", error);

  cusparseCheck(hipsparseDestroyDnVec(tempDesc));
  cusparseCheck(hipsparseDestroyDnVec(tmpDesc));
  cusparseCheck(hipsparseDestroySpMat(Adescriptor));

  cusparseCheck(hipsparseDestroy(cusparseHandle));

  cublasCheck(hipblasDestroy(cublasHandle));

  hipFree(temp);
  hipFree(A);
  hipFree(ARowPtr);
  hipFree(AColIndx);
  hipFree(tmp);
  hipFree(buffer);

  return 0;
}
